// cuda_hello.c



#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int main() {
    float *a = (float*)malloc(sizeof(float) * N);
    float *b = (float*)malloc(sizeof(float) * N);
    float *out = (float*)malloc(sizeof(float) * N);
    for (int i = 0; i < N; ++i) {
        a[i] = 1.0f; b[i] = 2.0f;
    }
    vector_add<<<1,1>>>(out,a,b,N);
    for (int i = 10; i < 20; ++i) {
        printf("%f\n", out[i]);
    }
    return 0;
}
